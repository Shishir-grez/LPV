#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <iomanip> // For setprecision
#include <omp.h> // For omp_get_wtime

using namespace std;

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Kernel for tiled matrix multiplication
__global__ void matrixMulKernel(float* A, float* B, float* C, int M, int N, int K) {
    const int TILE_SIZE = 16;
    __shared__ float As[16][16];
    __shared__ float Bs[16][16];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load tiles into shared memory
        if (row < M && t * TILE_SIZE + threadIdx.x < K)
            As[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && t * TILE_SIZE + threadIdx.y < K)
            Bs[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Compute partial sum
        for (int i = 0; i < TILE_SIZE; ++i)
            sum += As[threadIdx.y][i] * Bs[i][threadIdx.x];

        __syncthreads();
    }

    if (row < M && col < N)
        C[row * N + col] = sum;
}

// Sequential matrix multiplication for verification
void sequentialMatrixMul(const vector<float>& A, const vector<float>& B, vector<float>& C, int M, int N, int K) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < K; ++k) {
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

// Function to generate random matrix
vector<float> generateRandomMatrix(int rows, int cols) {
    vector<float> mat(rows * cols);
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<float> dis(0.0f, 10.0f);
    for (int i = 0; i < rows * cols; ++i) {
        mat[i] = dis(gen);
    }
    return mat;
}

int main() {
    int M = 1024;  // Matrix A: 1024 rows
    int K = 1024;  // Matrix A: 1024 columns, Matrix B: 1024 rows
    int N = 1024;  // Matrix B: 1024 columns
    
    cout << "Using hardcoded matrix dimensions: A(" << M << "x" << K << ") * B(" << K << "x" << N << ") = C(" << M << "x" << N << ")" << endl;
    
    // Always use random matrices
    cout << "Generating random matrices..." << endl;
    vector<float> A = generateRandomMatrix(M, K);
    vector<float> B = generateRandomMatrix(K, N);
    vector<float> C(M * N), C_seq(M * N);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, M * K * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, K * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C, M * N * sizeof(float)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_A, A.data(), M * K * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B.data(), K * N * sizeof(float), hipMemcpyHostToDevice));

    // Set up kernel launch parameters
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Launch kernel and measure time
    CUDA_CHECK(hipEventRecord(start));
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float cuda_time_ms;
    CUDA_CHECK(hipEventElapsedTime(&cuda_time_ms, start, stop));

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(C.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Sequential matrix multiplication for timing
    double seq_start = omp_get_wtime();
    sequentialMatrixMul(A, B, C_seq, M, N, K);
    double seq_end = omp_get_wtime();
    double seq_time = seq_end - seq_start;

    // Output result matrix (trimmed if large)
    cout << "\nResult Matrix C (first 5 elements, row-major):\n";
    for (int i = 0; i < min(5, M * N); ++i) {
        cout << C[i] << " ";
    }
    if (M * N > 5) cout << "...";
    cout << endl;

    // Output execution times and stats
    cout << "\nExecution Times:\n";
    cout << "CUDA Matrix Multiplication: " << fixed << setprecision(6) << cuda_time_ms / 1000.0 << " seconds\n";
    cout << "Sequential Matrix Multiplication: " << seq_time << " seconds\n";
    cout << "Speedup (Sequential / CUDA): " << seq_time / (cuda_time_ms / 1000.0) << "x\n";

    // Free device memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}