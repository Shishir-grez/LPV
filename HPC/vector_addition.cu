#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <iomanip>

using namespace std;

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Kernel for vector addition
__global__ void vectorAddKernel(float* A, float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

// Sequential vector addition for verification
void sequentialVectorAdd(const vector<float>& A, const vector<float>& B, vector<float>& C, int N) {
    for (int i = 0; i < N; ++i) {
        C[i] = A[i] + B[i];
    }
}

// Function to generate random vector
vector<float> generateRandomVector(int N) {
    vector<float> vec(N);
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<float> dis(0.0f, 10.0f);
    for (int i = 0; i < N; ++i) {
        vec[i] = dis(gen);
    }
    return vec;
}

int main() {
    int N;
    char choice;

    // Input vector length - Modified for automated testing
    N = 50000000;  // Using a large value for better timing comparison
    choice = 'y'; // Always use random vectors in Kaggle

    vector<float> A, B, C(N), C_seq(N);
    if (choice == 'y' || choice == 'Y') {
        A = generateRandomVector(N);
        B = generateRandomVector(N);
    } else {
        A.resize(N);
        B.resize(N);
        cout << "Enter " << N << " elements for vector A:\n";
        for (int i = 0; i < N; ++i) cin >> A[i];
        cout << "Enter " << N << " elements for vector B:\n";
        for (int i = 0; i < N; ++i) cin >> B[i];
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C, N * sizeof(float)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_A, A.data(), N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B.data(), N * sizeof(float), hipMemcpyHostToDevice));

    // Set up kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Launch kernel and measure time
    CUDA_CHECK(hipEventRecord(start));
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float cuda_time_ms;
    CUDA_CHECK(hipEventElapsedTime(&cuda_time_ms, start, stop));

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(C.data(), d_C, N * sizeof(float), hipMemcpyDeviceToHost));

    // Sequential vector addition for timing
    double seq_start = omp_get_wtime();
    sequentialVectorAdd(A, B, C_seq, N);
    double seq_end = omp_get_wtime();
    double seq_time = seq_end - seq_start;

    // Output result vector (trimmed if large)
    cout << "\nResult Vector C (first 5 elements):\n";
    for (int i = 0; i < min(5, N); ++i) {
        cout << C[i] << " ";
    }
    if (N > 5) cout << "...";
    cout << endl;

    // Output execution times and stats
    cout << "\nExecution Times:\n";
    cout << "CUDA Vector Addition: " << fixed << setprecision(6) << cuda_time_ms / 1000.0 << " seconds\n";
    cout << "Sequential Vector Addition: " << seq_time << " seconds\n";
    cout << "Speedup (Sequential / CUDA): " << seq_time / (cuda_time_ms / 1000.0) << "x\n";

    // Free device memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}